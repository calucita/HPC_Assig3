#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> 
#include <sys/time.h>

#define MAX_ITER 10000
#define NODES 1000 // boundaries included



__global__ void kernel_jacobi(double *u, double *u_old, double *f, int N){

	
	int i,j;
	double h;

	h = 2.0/((double)N-1);

	i = blockIdx.y * blockDim.y + threadIdx.y+1;
	j = blockIdx.x * blockDim.x + threadIdx.x+1;

	// computing solution
	
	if (j < N-1 && i < N-1){
		
		u[i+j*N] = 0.25 * ( u_old[(i-1)+j*N] + u_old[(i+1)+j*N] + \
				    u_old[i+(j-1)*N] + u_old[i+(j+1)*N] + \
			            h*h*f[i+j*N] );	
	}
	
}

int main(int argc, char *argv[])
{

	struct timeval t1, t2;
	gettimeofday(&t1, NULL);	

	int N = NODES;
	int sizeXGrid = 1;
	int sizeXBlock = N-2; 

	if (argc == 2)
	{	
		N = atoi(argv[1]);
		if (N <= 18){ 
			sizeXBlock = N-2; 
			sizeXGrid = 1;
		} else {	
			sizeXBlock = 16; // each block can be at most (max is 512!!)
			sizeXGrid = ((N-2)+sizeXBlock-1)/sizeXBlock;
		}
	}

	// variables declaration
	int i, j, k, max_iter;
	double *u_h, *u_old_h, *f_h;
	double *u_d, *u_old_d, *f_d;
	double conv, *temp;


	dim3 DimGrid(sizeXGrid,sizeXGrid);
	dim3 DimBlock(sizeXBlock,sizeXBlock); // 484 threads per block 

	max_iter = MAX_ITER;
	
	// initializing stopwatches
	StopWatchInterface *timeKer;
	sdkCreateTimer(&timeKer);

	StopWatchInterface *timeMem1; //time to allocate to host
	sdkCreateTimer(&timeMem1);

	StopWatchInterface *timeMem2; //time to copy to device
	sdkCreateTimer(&timeMem1);

	StopWatchInterface *timeMem3; //time to copy to host
	sdkCreateTimer(&timeMem1);

	// allocating solution and forcing term in the host 
	f_h     = (double *)malloc(N*N * sizeof(double));
	u_h     = (double *)malloc(N*N * sizeof(double));
	u_old_h = (double *)malloc(N*N * sizeof(double));	

	
	//initialinzing solution and forcing term
	conv = 2.0/((double)N);
	for(j=0; j<N; j++){
		for(i=0; i<N; i++)
		{		
			if( conv*i-1.0 > 0.0 && conv*i-1.0 < 1.0/3.0 &&\
		   conv*j-1.0 > -2.0/3.0 && conv*j-1.0 < -1.0/3.0){			
				f_h[i+j*N]=200;
			}else {f_h[i+j*N]=0;}

			u_h[i+j*N] = 0;
			u_old_h[i+j*N] = 0;
			// boundary conditions
			if(i == 0 || j == N-1 || i == N-1){ 
				u_h[i+j*N] = 20;
				u_old_h[i+j*N] = 20;
			}
		} 
	}
	
	sdkStartTimer(&timeMem1);
	// allocating solution and forcing term in the device
	hipMalloc((void**)&f_d,N*N * sizeof(double));
	hipMalloc((void**)&u_d,N*N * sizeof(double));
	hipMalloc((void**)&u_old_d,N*N * sizeof(double));
	sdkStopTimer(&timeMem1);

	sdkStartTimer(&timeMem2);
	// copying from host to device
	hipMemcpy(u_d,u_h, N*N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(u_old_d,u_old_h, N*N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(f_d,f_h, N*N*sizeof(double),hipMemcpyHostToDevice);
	sdkStopTimer(&timeMem2);

	// calling kernel and taking time
	k = 1;
	sdkStartTimer(&timeKer);
	while (k < max_iter){
		temp = u_d;
		u_d = u_old_d;
		u_old_d = temp;
		kernel_jacobi<<< DimGrid, DimBlock >>>(u_d, u_old_d, f_d, N);
		hipDeviceSynchronize();		
		k++;
	}
	sdkStopTimer(&timeKer);
	

	// copying from device to host
	sdkStartTimer(&timeMem3);
	hipMemcpy(u_h,u_d, N*N *sizeof(double),hipMemcpyDeviceToHost);
	sdkStopTimer(&timeMem3);

	// print solution

	/*FILE * fp;

   	fp = fopen ("solution2.txt", "w+");

	for(j=0; j<N; j++){
		for(i=0; i< N; i++){
			fprintf(fp, "%lf ",u_h[i+j*N]);
		}
		fprintf(fp, "\n");
	}
   
   	fclose(fp);*/

	// freeing memory	
	free(u_old_h);
	free(u_h);
	free(f_h);
	hipFree(u_old_d);
	hipFree(u_d);
	hipFree(f_d);	

	// print time
	double tK = sdkGetTimerValue(&timeKer);
	double tM1 = sdkGetTimerValue(&timeMem1);
	double tM2 = sdkGetTimerValue(&timeMem2);
	double tM3 = sdkGetTimerValue(&timeMem3);

	printf("Kernel time: %lf \n", tK/1e3);
	printf("Memory time: %lf \n", (tM1 + tM2 + tM3)/1e3);

	//printf("Block size: %i x %i \n", sizeXBlock,sizeXBlock);
	//printf("Grid size: %i x %i \n", sizeXGrid, sizeXGrid);

	double gputime = (tK + tM1 + tM2 + tM3)/1e3;

	gettimeofday(&t2, NULL);
	double  walltime = t2.tv_sec - t1.tv_sec + (t2.tv_usec -t1.tv_usec) / 1.e6;
	double cputime = walltime - gputime;
	printf("Wall time: %lf \n", walltime);
	printf("CPU time: %lf \n", cputime);

	double kerneltime = tK/max_iter;
	double flops = ((N*N*5)/1e9)/(kerneltime/1e3);
	printf("Performance [GFlop/s]: %lf \n", flops);
	
	
	return 0;

}
