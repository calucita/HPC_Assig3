#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <sunperf.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <sys/time.h>
#include <hipblas.h>


#define min(a,b)(((a)<(b))?(a):(b))
#define SIZEXBLOCK 32 
#define SIZEYBLOCK 32

#ifndef MATMULT_LIB_H
#define MATMULT_LIB_H
extern "C" {
#include <cblas.h>

__global__ void gpu5(int m, int n, int k, double *a, double *b, double *c){
	double sum=0;
	int globalThreadIdx=blockIdx.x*blockDim.x+threadIdx.x;
	int globalThreadIdy=blockIdx.y*blockDim.y+threadIdx.y;
	int i,j;
	__shared__ double A_s[SIZEXBLOCK][SIZEXBLOCK];
	__shared__ double B_s[SIZEXBLOCK][SIZEXBLOCK];
	if (globalThreadIdx < m && globalThreadIdy < n) {
		for(j=0;j<k; j+=SIZEXBLOCK){
			A_s[threadIdx.y][threadIdx.x]=a[globalThreadIdy+(j+threadIdx.x)*k];
			B_s[threadIdx.x][threadIdx.y]=b[globalThreadIdx*n+j+threadIdx.y];
			__syncthreads();
			for (i =0; i< SIZEXBLOCK; i++){
				if (j+i<k){
				//	if(m%SIZEXBLOCK==0 || n/SIZEXBLOCK == 0 || j<k-SIZEXBLOCK){
						sum+=A_s[threadIdx.y][i]*B_s[threadIdx.x][i];
//					} else if( j<k-SIZEXBLOCK ){
//						sum+=A_s[threadIdx.y][i]*b[globalThreadIdx*n+i+j];
				//	}else if ((threadIdx.y < n%SIZEXBLOCK && j>k-SIZEXBLOCK)){
				//		sum+=A_s[threadIdx.y][i]*b[globalThreadIdx*n+i+j];
					//	printf("I actually do something \n");
		//				sum+=a[globalThreadIdx*k+j+i]*b[globalThreadIdy+(i+j)*n];
					//}//else {printf("oh, hai, I'm skipping around! \n");}
				}
			}
			__syncthreads();
		}
		c[globalThreadIdy+globalThreadIdx*n]=sum;	
	}
}


void matmult_gpu5(int m, int n, int k, double **A, double **B, double **C){
	int sizeXBlock = SIZEXBLOCK;
	int sizeXGrid = (m+sizeXBlock-1)/sizeXBlock;
	int sizeYBlock = SIZEYBLOCK;
	int sizeYGrid =  (n+sizeYBlock-1)/sizeYBlock;
	double *a_d;
	double *b_d;
	double *c_d;

	dim3 DimGrid(sizeXGrid,sizeYGrid);
	dim3 DimBlock(sizeXBlock, sizeYBlock);

	checkCudaErrors(hipMalloc((void**)&a_d,m*k*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&b_d,k*n*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&c_d,m*n*sizeof(double)));
	checkCudaErrors(hipMemcpy(a_d,A[0], m*k*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d,B[0], k*n*sizeof(double),hipMemcpyHostToDevice));

	gpu5<<< DimGrid, DimBlock>>>(m,n,k,a_d,b_d,c_d);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(C[0],c_d, m*n*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	checkCudaErrors(hipGetLastError());
}

__global__ void gpu4(int m, int n, int k, double *a, double *b, double *c){
	double sum=0;
	int globalThreadIdx=blockIdx.x*blockDim.x+threadIdx.x;
	int globalThreadIdy=blockIdx.y*blockDim.y+threadIdx.y;
	int i,j;
	__shared__ double A_s[SIZEXBLOCK][SIZEXBLOCK];
	if (globalThreadIdx < m && globalThreadIdy < n) {
		for(j=0;j<k; j+=SIZEXBLOCK){
			A_s[threadIdx.y][threadIdx.x]=a[globalThreadIdy+(j+threadIdx.x)*k];
			__syncthreads();
			for (i =0; i< SIZEXBLOCK; i++){
				if (j+i<k){
					sum+=A_s[threadIdx.y][i]*b[globalThreadIdx*n+i+j];
				}
			}
			__syncthreads();
		}
		c[globalThreadIdy+globalThreadIdx*n]=sum;	
	}
}


void matmult_gpu4(int m, int n, int k, double **A, double **B, double **C){
	int sizeXBlock = SIZEXBLOCK;
	int sizeXGrid = (m+sizeXBlock-1)/sizeXBlock;
	int sizeYBlock = SIZEYBLOCK;
	int sizeYGrid =  (n+sizeYBlock-1)/sizeYBlock;
	double *a_d;
	double *b_d;
	double *c_d;

	dim3 DimGrid(sizeXGrid,sizeYGrid);
	dim3 DimBlock(sizeXBlock, sizeYBlock);

	checkCudaErrors(hipMalloc((void**)&a_d,m*k*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&b_d,k*n*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&c_d,m*n*sizeof(double)));
	checkCudaErrors(hipMemcpy(a_d,A[0], m*k*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d,B[0], k*n*sizeof(double),hipMemcpyHostToDevice));

	gpu4<<< DimGrid, DimBlock>>>(m,n,k,a_d,b_d,c_d);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(C[0],c_d, m*n*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	checkCudaErrors(hipGetLastError());
}

__global__ void gpu3(int m, int n, int k, double *a, double *b, double *c)
{
	int j,nlim=n/4;
	int Idx=blockIdx.x*blockDim.x+threadIdx.x;
	int Idy=blockIdx.y*blockDim.y+threadIdx.y;
	double sum0=0, sum1=0, sum2=0, sum3=0;
        if (Idx < n && Idy < nlim) {
		for(j=0;j<k;j++){
			sum0+=a[Idy*k+j]*b[Idx+j*n];
			sum1+=a[Idy*k+j]*b[Idx+nlim+j*n];
			sum2+=a[Idy*k+j]*b[Idx+2*nlim+j*n];
			sum3+=a[Idy*k+j]*b[Idx+3*nlim+j*n];
		}
		c[(Idx*n)+(Idy)]=sum0;
		c[(Idx*n)+(Idy+nlim)]=sum1;
		c[(Idx*n)+(Idy+2*nlim)]=sum2;
		c[(Idx*n)+(Idy+3*nlim)]=sum3;
	}
}


void matmult_gpu3(int m, int n, int k, double **A, double **B, double **C)
{
	int sizeXBlock = SIZEXBLOCK/4;
	int sizeXGrid = (m+sizeXBlock-1)/sizeXBlock;
	int sizeYBlock = SIZEYBLOCK;
	int sizeYGrid =  (n+sizeYBlock-1)/sizeYBlock;
	double *a_d;
	double *b_d;
	double *c_d;

	dim3 DimGrid(sizeXGrid,sizeYGrid);
	dim3 DimBlock(sizeXBlock, sizeYBlock);
	checkCudaErrors(hipMalloc((void**)&a_d,m*k*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&b_d,k*n*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&c_d,m*n*sizeof(double)));
	checkCudaErrors(hipMemcpy(a_d,A[0], m*k*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d,B[0], k*n*sizeof(double),hipMemcpyHostToDevice));	
	checkCudaErrors(hipGetLastError());
	gpu3<<< DimGrid, DimBlock >>>(m,n,k,a_d,b_d,c_d);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(C[0],c_d, m*n*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}

__global__ void gpu2(int m, int n, int k, double *a, double *b, double *c)
{
	int j;
	double sum1=0;
	double sum2=0;
	int globalThreadIdx=blockIdx.x*blockDim.x+threadIdx.x;
	int globalThreadIdy=blockIdx.y*blockDim.y+threadIdx.y;	
	if (globalThreadIdx < m && globalThreadIdy < n/2) {
		for(j=0;j<k;j++){
			sum1+=a[globalThreadIdy*k+j]*b[globalThreadIdx+j*n];
			sum2+=a[globalThreadIdy*k+j]*b[(globalThreadIdx+n/2)+j*n];
		}
		c[globalThreadIdy+globalThreadIdx*n]=sum1;
		c[globalThreadIdy+n/2+globalThreadIdx*n]=sum2;		
	}
}



void matmult_gpu2(int m, int n, int k, double **A, double **B, double **C)
{
int sizeXBlock = 32;
	int sizeXGrid = (m+sizeXBlock-1)/sizeXBlock;
	int sizeYBlock = 32;
	int sizeYGrid =  (n+sizeYBlock-1)/sizeYBlock;
	double *a_d;
	double *b_d;
	double *c_d;

	dim3 DimGrid(sizeXGrid,sizeYGrid);
	dim3 DimBlock(sizeXBlock, sizeYBlock);

	checkCudaErrors(hipMalloc((void**)&a_d,m*k*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&b_d,k*n*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&c_d,m*n*sizeof(double)));
	checkCudaErrors(hipMemcpy(a_d,A[0], m*k*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d,B[0], k*n*sizeof(double),hipMemcpyHostToDevice));

	gpu2<<< DimGrid, DimBlock >>>(m,n,k,a_d,b_d,c_d);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(C[0],c_d, m*n*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}





__global__ void gpu1(int m, int n, int k, double *a, double *b, double *c)
{
	int j;
	double sum=0;
	int globalThreadIdx=blockIdx.x*blockDim.x+threadIdx.x;
	int globalThreadIdy=blockIdx.y*blockDim.y+threadIdx.y;	
	if (globalThreadIdx < m && globalThreadIdy < n) {
		for(j=0;j<k;j++){
			sum+=a[globalThreadIdy*k+j]*b[globalThreadIdx+j*n];
		}
		c[globalThreadIdx+globalThreadIdy*n]=sum;	
	}
}



void matmult_gpu1(int m, int n, int k, double **A, double **B, double **C)
{
	int sizeXBlock = 32;
	int sizeXGrid = (m+sizeXBlock-1)/sizeXBlock;
	int sizeYBlock = 32;
	int sizeYGrid =  (n+sizeYBlock-1)/sizeYBlock;

	double *a_d;
	double *b_d;
	double *c_d;

	dim3 DimGrid(sizeXGrid,sizeYGrid);
	dim3 DimBlock(sizeXBlock, sizeYBlock);

	checkCudaErrors(hipMalloc((void**)&a_d,m*k*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&b_d,k*n*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&c_d,m*n*sizeof(double)));
	checkCudaErrors(hipMemcpy(a_d,A[0], m*k*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d,B[0], k*n*sizeof(double),hipMemcpyHostToDevice));

	gpu1<<< DimGrid, DimBlock >>>(m,n,k,a_d,b_d,c_d);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(C[0],c_d, m*n*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}

void matmult_lib(int m, int n, int k, double **A, double **B, double **C){
	double alpha, beta;
	alpha = 1.0; beta = 0.0;
	cblas_dgemm(CblasColMajor,CblasNoTrans,CblasNoTrans, n, m, k, alpha, B[0], n, A[0], k, beta, C[0], n);		
}

void matmult_gpulib(int m, int n, int k, double **A, double **B, double **C){
	double alpha, beta,*A_d, *B_d, *C_d;
	alpha = 1.0; beta = 0.0;
	checkCudaErrors(hipMalloc((void**)&A_d, (m*k*sizeof(double*))));
	checkCudaErrors(hipMalloc((void**)&B_d, (k*n*sizeof(double*))));
	checkCudaErrors(hipMalloc((void**)&C_d, (m*n*sizeof(double*)))); 
	checkCudaErrors(hipMemcpy(A_d, A[0], (m*k*sizeof(double)), hipMemcpyHostToDevice));	
	checkCudaErrors(hipMemcpy(B_d, B[0], (k*n*sizeof(double)), hipMemcpyHostToDevice));	
	
	hipblasDgemm('n', 'n', n, m, k, alpha, B_d, n, A_d, k, beta, C_d, n);
		
	checkCudaErrors(hipMemcpy(C[0],C_d, (m*n*sizeof(double)), hipMemcpyDeviceToHost));	
	checkCudaErrors(hipFree(A_d));
	checkCudaErrors(hipFree(B_d));
	checkCudaErrors(hipFree(C_d));
}

void matmult_nat(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
			for ( t = 0; t < k ; t++){
				C[i][j] = C[i][j]+ A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_mnk(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
			for ( t = 0; t < k ; t++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_mkn(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}
	for ( i = 0; i < m ; i++){
		for ( t = 0; t < k ; t++){
			for ( j = 0; j < n; j++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_nmk(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}for ( j = 0; j < n; j++){
		for ( i = 0; i < m ; i++){
			for ( t = 0; t < k ; t++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_nkm(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}for ( j = 0; j < n; j++){
		for ( t = 0; t < k ; t++){
			for ( i = 0; i < m ; i++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_knm(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}for ( t = 0; t < k ; t++){
		for ( j = 0; j < n; j++){
			for ( i = 0; i < m ; i++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_kmn(int m, int n, int k, double **A, double **B, double **C){
	int i,j,t;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}for ( t = 0; t < k ; t++){
		for ( i = 0; i < m ; i++){
			for ( j = 0; j < n; j++){
				C[i][j] += A[i][t]*B[t][j];
			}	
		}
	}
}

void matmult_blk(int m, int n, int l, double **A, double **B, double **C, int bb){
	int i,j,k,jj,kk;
	for ( i = 0; i < m ; i++){
		for ( j = 0; j < n; j++){
			C[i][j]=0;
		}
	}
	double r;
	//size of block from number of elements
	bb=sqrt(bb);
	//blocked multiplication
	for(kk=0;kk<l;kk+=bb){
		for(jj=0;jj<n;jj+=bb){
			for(i=0;i<m;i++){
				for(k=kk;k<min(kk+bb,l);k++){
					r=A[i][k];
					for(j=jj;j<min(jj+bb,n);j++){
						C[i][j]+=r*B[k][j];
					}
				}
			}
		}
	}
}

}



#endif
